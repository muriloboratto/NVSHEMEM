#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>

__global__ void simple_shift(int *destination) 
{
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int main(int argc, char **argv) 
{
    int mype_node, msg;
    hipStream_t stream;

    nvshmem_init();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    int *destination = (int *) nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    return 0;
}